#include "hip/hip_runtime.h"
#include "../common/book.h"


#define 	N	10

__global__ void add(int *a,int *b,int *c) {
	int tid = threadIdx.x;
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
}

int main(void) {

	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// fill the arrays ‘a’ and ‘b’ on the CPU
	for (int i=0; i<N; i++) {
		a[i] = i;
		b[i] = i * i;
	}

	// copy the arrays ‘a’ and ‘b’ to the GPU
	HANDLE_ERROR(hipMemcpy(dev_a,a,N * sizeof(int),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b,b,N * sizeof(int),hipMemcpyHostToDevice));
	add<<<1,N>>>(dev_a, dev_b, dev_c);

	// copy the array ‘c’ back from the GPU to the CPU
	HANDLE_ERROR(hipMemcpy(c,dev_c,N * sizeof(int),hipMemcpyDeviceToHost));
	// display the results
	for (int i=0; i<N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	// free the memory allocated on the GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
