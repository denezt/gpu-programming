#include "hip/hip_runtime.h"
// An intermediate level GPU Program

#include <stdio.h>
#include <stdlib.h>
#include "../common/book.h"


__global__ void vectorAdd(int *a, int *b, int *c, int n){
	// Calculate Index Thread
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// Make sure we stay in-bounds
	if (tid < n){
		// Vector Add
		c[tid] = a[tid] + b[tid];
	}
}

int main(){
	// Number of elements
	int n = 1 << 20;

	// Host Pointers
	int *h_a;
	int *h_b;
	int *h_c;

	// Device Pointers
	int *d_a;
	int *d_b;
	int *d_c;

	size_t bytes = n * sizeof(int);

	// Allocate memory on host side
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	for (int i = 0; i < n; i++){
		h_a[i] = 1;
		h_b[i] = 2;
	}

	// Allocate memory on device side
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);
	
	// Init block and grid size
	int block_size = 1024;
	int grid_size = (int)ceil((float) n /block_size);
	printf("Grid size is %d\n", grid_size);

	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	vectorAdd<<<grid_size, block_size>>>(d_a, d_b, d_c, n);

	hipMemcpy(h_c,d_c,bytes, hipMemcpyHostToDevice);

	for(int i =0; i < n; i++){
		if (h_c[i] != 3){
			printf("ERROR!\n");
			break;
		}
	}

	printf("Completed Successfully!\n");

	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
