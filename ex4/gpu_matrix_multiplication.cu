#include "hip/hip_runtime.h"
#include "../common/book.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void matrixMul(int* m, int* n, int* p, int size){
	// Calculate Row and Column
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;

	// Partial Sum Element
	int p_sum = 0;
	for (int i = 0; i < size; i++) {
		p_sum += m[row * size + i] * n[i * size + column];
	}
	p[row * size + column] = p_sum;
}

void matrixMul_seq(int *m, int *n, int *p, int size){
	for (int i = 0; i < size; i++){
		for (int j = 0; j < size; j++){
			for (int k = 0; k < size; k++){
				p[i * size + j] += m[i * size + k] * n[k * size + j];
			}
		}
	}
}

int main() {
	int n = 1 << 10; // equals 1024 or 2^10
	printf("Square matrix of size %d\n", n);

	/**
	 * We should always start with our
	 * host side of the code first
	 * and compile the code.
	 */

	// Host Matrix m,n,p
	int* h_m;
	int* h_n;
	int* h_p;
	int* h_p_seq;

	// Device Matrix m,n,p
	int* d_m;
	int* d_n;
	int* d_p;

	// Matrix Sizing n times n elements (integers)
	size_t bytes = n * n * sizeof(int);

	// Allocate memory on host side
	h_m = (int*)malloc(bytes);
	h_n = (int*)malloc(bytes);
	h_p = (int*)malloc(bytes);
	h_p_seq = (int*)malloc(bytes);

	// Initialize matrix m, n, p
	for(int i = 0; i < n; i++){
		for (int j = 0; j < n; j++){
			h_m[i*n + j] = rand() % 1024;
			h_n[i*n + j] = rand() % 1024;
		}
	}

	// Allocate memory on device side
	hipMalloc(&d_m, bytes);
	hipMalloc(&d_n, bytes);
	hipMalloc(&d_p, bytes);

	// Copy data to the device
	hipMemcpy(d_m, h_m, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_n, h_n, bytes, hipMemcpyHostToDevice);
	
	int threads_per_block = 16;
	// Cuda specific parameter type (x,y,z) structs
	dim3 block_size(threads_per_block, threads_per_block);
	dim3 grid_size(n / block_size.x, n / block_size.y);

	printf("Grid size X: %d, Grid size Y: %d\n", grid_size.x, grid_size.y);
	
	matrixMul<<<grid_size,block_size>>>(d_m, d_n,d_p,n);
	matrixMul_seq(h_m, h_n, h_p_seq,n);

	// Copy to the host pointer from the device pointer
	hipMemcpy(h_p, d_p, bytes, hipMemcpyDeviceToHost);

	// Display outputs
	printf("CPU Computation: %d\n",h_p_seq[0]);
	printf("GPU Computation: %d\n",h_p[0]);

	// Free allocated host memory
	free(h_m);
	free(h_n);
	free(h_p);
	free(h_p_seq);

	// Free CUDA allocated memory
	hipFree(&d_m);
	hipFree(&d_n);
	hipFree(&d_p);

	return 0;
}
